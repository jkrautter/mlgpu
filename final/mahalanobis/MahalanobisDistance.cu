#include "hip/hip_runtime.h"
#include <hipblas.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/random.h>
#include <thrust/sequence.h>

#include <stdio.h>
#include <iostream>
#include <math.h> 

#include "Utilities.cuh"
#include "TimingGPU.cuh"

float computeRangeBetweenVectors(float* a, float* b, float*  inverse_covariance_matrix,  int n)
{
    thrust::device_vector<float> d_a(n);
    thrust::device_vector<float> d_b(n);
    thrust::device_vector<float> d_inv_cov(n*n);
    thrust::device_vector<float> d_tmp_result(n);
    thrust::device_vector<float> d_distance;
	
    for (size_t i = 0; i < n; i++)   d_a[i] = a[i];
    for (size_t i = 0; i < n; i++)   d_b[i] = b[i];
    for (size_t i = 0; i < n*n; i++) d_inv_cov[i] = inverse_covariance_matrix[i];

    float alpha = 1.f;
    float beta = 0.f;
   
    // --- cuBLAS handle creation
    hipblasHandle_t handle;
    cublasSafeCall(hipblasCreate(&handle));

    cublasSafeCall(
	hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 1, n, n,
			    &alpha,
		            thrust::raw_pointer_cast(d_a.data()), n,
		            thrust::raw_pointer_cast(d_inv_cov.data()), n,
			    &beta,
		            thrust::raw_pointer_cast(d_tmp_result.data()), 1));

     cublasSafeCall(
	hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 1, n, 1, 
		            &alpha,
		            thrust::raw_pointer_cast(d_tmp_result.data()), 1,
		            thrust::raw_pointer_cast(d_inv_cov.data()), n,
			    &beta,
		            thrust::raw_pointer_cast(d_distance.data()), 1));

   float result = sqrt(d_distance[0]);
   return result;
}



thrust::device_vector<float> computeInverseMatrix(float* mat, int n){
//float* computeInverseMatrix(float* mat, int n){

    thrust::device_vector<float> d_matrix(n*n);
    for (size_t i = 0; i < n*n; i++) d_matrix[i] = mat[i];

    // --- cuBLAS handle creation
    hipblasHandle_t handle;
    cublasSafeCall(hipblasCreate(&handle));
 
    thrust::device_vector<int> d_pivot_vector(n);
    thrust::device_vector<int> d_info(1);
    thrust::device_vector<float> d_inv_matrix(n*n);

    float* ptr_matrix = thrust::raw_pointer_cast(d_matrix.data());
    int* ptr_pivot = thrust::raw_pointer_cast(d_pivot_vector.data());
    int* ptr_info = thrust::raw_pointer_cast(d_info.data());
    float* ptr_inv_matrix = thrust::raw_pointer_cast(d_inv_matrix.data());

    printf("Starting Matrix Inversion");

    cublasSafeCall(
	hipblasSgetrfBatched(handle, n, &ptr_matrix, n, ptr_pivot, ptr_info, 1));
    hipDeviceSynchronize();
    cublasSafeCall(
	hipblasSgetriBatched(handle, n,(const float **) &ptr_matrix, n, ptr_pivot, &ptr_inv_matrix, n, ptr_info, 1));
    hipDeviceSynchronize();

    return d_inv_matrix;
//    float *result;
//    result = (float *)malloc(n*sizeof(float));
//    thrust::copy(d_inv_matrix.begin(), d_inv_matrix.end(), result);
//    return result;
}


/*************************************/
/* CONVERT LINEAR INDEX TO ROW INDEX */
/*************************************/
template <typename T>
struct linear_index_to_row_index : public thrust::unary_function<T,T> {
	
	T Ncols; // --- Number of columns
  
	__host__ __device__ linear_index_to_row_index(T Ncols) : Ncols(Ncols) {}

	__host__ __device__ T operator()(T i) { return i / Ncols; }
};

//thrust::device_vector<float> computeCovarianceMatrix(int Nsamples, int NX, thrust::device_vector<float> d_X)
float* computeCovarianceMatrix(int Nsamples, int NX, thrust::device_vector<float> d_X)
{	
    // --- cuBLAS handle creation
    hipblasHandle_t handle;
    cublasSafeCall(hipblasCreate(&handle));

	/*************************************************/
    /* CALCULATING THE MEANS OF THE RANDOM VARIABLES */
	/*************************************************/
    // --- Array containing the means multiplied by Nsamples
	thrust::device_vector<float> d_means(NX);

	thrust::device_vector<float> d_ones(Nsamples, 1.f);

    float alpha = 1.f / (float)Nsamples;
    float beta  = 0.f;
    cublasSafeCall(
	hipblasSgemv(handle, HIPBLAS_OP_T, Nsamples, NX, 
			    &alpha, 
			    thrust::raw_pointer_cast(d_X.data()), Nsamples, 
                            thrust::raw_pointer_cast(d_ones.data()), 1,
 			    &beta,
			    thrust::raw_pointer_cast(d_means.data()), 1));
	
	/**********************************************/
    /* SUBTRACTING THE MEANS FROM THE MATRIX ROWS */
	/**********************************************/
	thrust::transform(
                d_X.begin(), d_X.end(),
                thrust::make_permutation_iterator(
                        d_means.begin(),
                        thrust::make_transform_iterator(thrust::make_counting_iterator(0), linear_index_to_row_index<int>(Nsamples))),
                d_X.begin(),
				thrust::minus<float>());	
	
	/*************************************/
    /* CALCULATING THE COVARIANCE MATRIX */
	/*************************************/
    thrust::device_vector<float> d_cov(NX * NX);

    alpha = 1.f;
    cublasSafeCall(
	hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, NX, NX, Nsamples,
			    &alpha,
		            thrust::raw_pointer_cast(d_X.data()), Nsamples,
			    thrust::raw_pointer_cast(d_X.data()), Nsamples,
			    &beta, 
			    thrust::raw_pointer_cast(d_cov.data()), NX));

	// --- Final normalization by Nsamples - 1
	thrust::transform(
                d_cov.begin(), d_cov.end(),
                thrust::make_constant_iterator((float)(Nsamples-1)),
                d_cov.begin(),
				thrust::divides<float>());	

	//return d_cov;
    float *result;
    result = (float *)malloc(NX*sizeof(float));
    thrust::copy(d_cov.begin(), d_cov.end(), result);
    return result;

}


/********/
/* MAIN */
/********/
int main()
{
    const int Nsamples = 3;		// --- Number of realizations for each random variable (number of rows of the X matrix)
    const int NX	= 2;		// --- Number of random variables (number of columns of the X matrix)

	// --- Random uniform integer distribution between 10 and 99
    thrust::default_random_engine rng;
    thrust::uniform_int_distribution<int> dist(10, 99);

    // --- Matrix allocation and initialization
    thrust::device_vector<float> d_X(Nsamples * NX);

//    for (size_t i = 0; i < d_X.size(); i++) d_X[i] = (float)dist(rng);

    d_X[0]=1.0;
    d_X[1]=2.0;
    d_X[2]=2.0;
    d_X[3]=1.0;
    d_X[4]=4.0;
    d_X[5]=4.0;

    //thrust::device_vector<float> d_cov;
    thrust::device_vector<float> d_inv_cov;
    float* d_cov = computeCovarianceMatrix(3,2, d_X);
    d_inv_cov = computeInverseMatrix(d_cov, 2);
                
    for(int i = 0; i < NX * NX; i++) std::cout << d_inv_cov[i] << "\n";

    return 0;
}


