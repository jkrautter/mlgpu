#include <stdio.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <fcntl.h>
#include <unistd.h>
#include <math.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <inttypes.h>

#define BLOCKSIZE 32
#define NUM_BLOCKS 1875

__global__ void computeDistances(float *testimage, float *trainimages, float *dist, unsigned int width, unsigned int height) {
	unsigned int image_id = blockIdx.x * blockDim.x + threadIdx.x;
	float sum = 0.0f;
	for (unsigned int i = 0; i < width * height; i++) {
		float factor = testimage[i] - trainimages[image_id * width * height + i];
		sum += factor * factor;
	}
	dist[image_id] = sqrt(sum);
}

#define TRAIN_IMAGES_FILE "train-images-idx3-ubyte"
#define TRAIN_LABELS_FILE "train-labels-idx1-ubyte"

#define TEST_IMAGES_FILE "t10k-images-idx3-ubyte"
#define TEST_LABELS_FILE "t10k-labels-idx1-ubyte"

#define WIDTH 28
#define HEIGHT 28
#define NUM_TRAIN_IMAGES 60000
#define NUM_TEST_IMAGES 10000
#define K 10
#define NUM_CLASSIFICATIONS 10000

struct dist {
  float dist;
  unsigned char label;
  int i;
};

float train_img[NUM_TRAIN_IMAGES * WIDTH * HEIGHT];
uint8_t train_label[NUM_TRAIN_IMAGES];

float test_img[NUM_TEST_IMAGES * WIDTH * HEIGHT];
unsigned char test_label[NUM_TEST_IMAGES];

struct dist dists[NUM_TRAIN_IMAGES];

int dist_cmp_func(const void *a, const void *b)
{
  struct dist *da = (struct dist *)a;
  struct dist *db = (struct dist *)b;
  return da->dist - db->dist;
}

float euclid_dist(int a, int b)
{
  float sum = 0.0f;

  int i = 0;
  for(i = 0; i < WIDTH * HEIGHT; i++)
  {
    int factor = train_img[a * WIDTH * HEIGHT + i] - test_img[b * WIDTH * HEIGHT + i];
    sum += factor * factor;
  }

  return sqrt(sum);
}

void read_files()
{
  int img_fd, label_fd;

  img_fd = open(TRAIN_IMAGES_FILE, O_RDONLY);
  label_fd = open(TRAIN_LABELS_FILE, O_RDONLY);

  if(img_fd < 0 || label_fd < 0)
  {
    printf("Cannot open training files\n");
    exit(1);
  }

  // ignore headers
  read(img_fd, &train_img, 4 * sizeof(int));
  read(label_fd, &train_img, 2 * sizeof(int));

  int i, j;
  for(i = 0; i < NUM_TRAIN_IMAGES; i++)
  {
	  uint8_t val = 0;
	  for (j = 0; j < WIDTH * HEIGHT; j++) {
		  read(img_fd, &val, 1);
		  train_img[i*WIDTH*HEIGHT + j] = (float) val;
	  }

    read(label_fd, &train_label[i], 1);
  }

  close(img_fd);
  close(label_fd);

  img_fd = open(TEST_IMAGES_FILE, O_RDONLY);
  label_fd = open(TEST_LABELS_FILE, O_RDONLY);

  if(img_fd < 0 || label_fd < 0)
  {
    printf("cannot open test files");
    exit(1);
  }

  read(img_fd, &test_img, 4 * sizeof(int));
  read(label_fd, &test_img, 2 * sizeof(int));

  for(i = 0; i < NUM_TEST_IMAGES; i++)
  {
	  uint8_t val = 0;
	  for (j = 0; j < WIDTH * HEIGHT; j++) {
		  read(img_fd, &val, 1);
		  test_img[i*WIDTH*HEIGHT + j] = (float) val;
	  }
    read(label_fd, &test_label[i], 1);
  }

  close(img_fd);
  close(label_fd);
}

void write_images(int ref)
{
  int fd = open("img/ref.pgm", O_WRONLY | O_CREAT | O_TRUNC, 0644);
  char header[1024];

  snprintf(header, 1024, "P5\n%d %d 255\n", WIDTH, HEIGHT);
  uint8_t tmp_test_img[NUM_TEST_IMAGES * WIDTH * HEIGHT];
  uint8_t tmp_train_img[NUM_TRAIN_IMAGES * WIDTH * HEIGHT];
  for (unsigned int i = 0; i < NUM_TRAIN_IMAGES * WIDTH * HEIGHT; i++) {
  	tmp_train_img[i] = (uint8_t) train_img[i];
  }
  for (unsigned int i = 0; i < NUM_TEST_IMAGES * WIDTH * HEIGHT; i++) {
  	tmp_test_img[i] = (uint8_t) test_img[i];
  }
  write(fd, header, strlen(header));
  write(fd, tmp_test_img, NUM_TEST_IMAGES * WIDTH * HEIGHT);

  close(fd);

  int k;
  for(k = 0; k < K; k++)
  { 
    char filename[1024];
    snprintf(filename, 1024, "img/nearest_%d.pgm", k);
    fd = open(filename, O_WRONLY | O_CREAT | O_TRUNC, 0644);

    snprintf(header, 1024, "P5\n%d %d 255\n", WIDTH, HEIGHT);

    write(fd, header, strlen(header));
    write(fd, &tmp_train_img[dists[k].i * WIDTH * HEIGHT], WIDTH * HEIGHT);

    close(fd);
  }
}

int main(int argc, char **argv)
{
  printf("Reading files...\n");
  read_files();
  printf("Files read.\n");
  int parallel = 0;
  if (argc > 1 && argv[1][0] == 'p') {
	  printf("Selected parallel computation.\n");
	  parallel = 1;
  }
  hipSetDevice(1);
  int freqs[10];
  int num_correct = 0;

  int ref;

  float *d_dists;
  float *d_images;
  float *d_testimage;

  hipMalloc(&d_images, NUM_TRAIN_IMAGES*WIDTH*HEIGHT*sizeof(float));
  hipMalloc(&d_dists, NUM_TRAIN_IMAGES*sizeof(float));
  hipMalloc(&d_testimage, WIDTH*HEIGHT*sizeof(float));

  hipMemcpy(d_images, train_img, NUM_TRAIN_IMAGES*WIDTH*HEIGHT*sizeof(float), hipMemcpyHostToDevice);

  printf("Starting classifications...\n");
  clock_t begin = clock();
  for(ref = 0; ref < NUM_CLASSIFICATIONS; ref++)
  {
    int i;
    if (!parallel) {
    	for(i = 0; i < NUM_TRAIN_IMAGES; i++)
    	{
    		dists[i].dist = euclid_dist(i, ref);
    		dists[i].label = train_label[i];
    		dists[i].i = i;
    	}
    } else {
    	hipMemcpy(d_testimage, &test_img[ref*WIDTH*HEIGHT], WIDTH*HEIGHT*sizeof(float), hipMemcpyHostToDevice);
    	computeDistances<<<NUM_BLOCKS, BLOCKSIZE>>>(d_testimage, d_images, d_dists, WIDTH, HEIGHT);
    	float tmp_dists[NUM_TRAIN_IMAGES];
    	hipMemcpy(tmp_dists, d_dists, NUM_TRAIN_IMAGES*sizeof(float), hipMemcpyDeviceToHost);

    	for(i = 0; i < NUM_TRAIN_IMAGES; i++)
    	{
    		dists[i].dist = tmp_dists[i];
    		dists[i].label = train_label[i];
    		dists[i].i = i;
    	}
    }

    qsort(dists, NUM_TRAIN_IMAGES, sizeof(struct dist), dist_cmp_func);

    for(i = 0; i < 10; i++)
      freqs[i] = 0;

    for(i = 0; i < K; i++)
      freqs[dists[i].label]++;

    int max = 0;
    int max_i = 0;

    for(i = 0; i < 10; i++)
    {
      if(freqs[i] >= max)
      {
        max = freqs[i];
        max_i = i;
      }
    }

    //printf("Guessed label: %d (%.2f%% of %d nearest). Actual label is %d%s\n", max_i, (float)max * 100.0 / K, K, test_label[ref], max_i == test_label[ref] ? ": correct": "");
    if(max_i == test_label[ref])
      num_correct++;

    // write_images(ref); // for debugging
  }
  clock_t end = clock();
  printf("Classification finished, CPU-time: %f\n", (double) (end - begin) / CLOCKS_PER_SEC);
  hipFree(d_images);
  hipFree(d_dists);
  hipFree(d_testimage);
  printf("Accuracy: %.2f\n", (float)num_correct / NUM_CLASSIFICATIONS);

  return 0;
}
